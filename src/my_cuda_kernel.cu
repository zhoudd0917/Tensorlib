#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <iostream>

// Error handling for cuBLAS
#define CHECK_CUBLAS_ERROR(err)                        \
  if (err != HIPBLAS_STATUS_SUCCESS) {                  \
    std::cerr << "cuBLAS error: " << err << std::endl; \
    return;                                            \
  }

void scale_vector_with_cublas(float* x, int n, float alpha) {
  // copy to device
  float* d_x;
  hipMalloc(&d_x, n * sizeof(float));
  hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);

  hipblasHandle_t handle;
  hipblasStatus_t status = hipblasCreate(&handle);
  CHECK_CUBLAS_ERROR(status);

  status = hipblasSscal(handle, n, &alpha, d_x, 1);
  CHECK_CUBLAS_ERROR(status);

  hipblasDestroy(handle);

  // copy back to host
  hipMemcpy(x, d_x, n * sizeof(float), hipMemcpyDeviceToHost);

  hipFree(d_x);
}