#include "hip/hip_runtime.h"
#include "cublas_handler.cuh"

CublasHandler& CublasHandler::getInstance() {
  static CublasHandler instance;
  return instance;
}

void CublasHandler::add(const float* x, const float* y, float* z, size_t size) {
  hipblasHandle_t handle = getInstance().getHandle();
  float alpha = 1.0;
  checkCudaErrors(
      hipMemcpy(z, y, size * sizeof(float), hipMemcpyDeviceToDevice));
  checkCublasErrors(hipblasSaxpy(handle, size, &alpha, x, 1, z, 1));
}

void CublasHandler::sub(const float* x, const float* y, float* z, size_t size) {
  hipblasHandle_t handle = getInstance().getHandle();
  float alpha = -1.0f;

  checkCudaErrors(
      hipMemcpy(z, x, size * sizeof(float), hipMemcpyDeviceToDevice));
  checkCublasErrors(hipblasSaxpy(handle, size, &alpha, y, 1, z, 1));
}

void CublasHandler::multiply(const float* x, const float* y, float* z,
                             size_t size) {
  hipblasHandle_t handle = getInstance().getHandle();
  hipblasSideMode_t mode = HIPBLAS_SIDE_LEFT;

  int m = size;    // Number of rows
  int n = 1;       // Single column (vector)
  int lda = size;  // Leading dimension of x (stride between rows)
  int incx = 1;    // Stride of y (scalar/vector elements)
  int ldc = size;  // Leading dimension of output z

  // Perform element-wise multiplication: z = diag(y) * x
  checkCublasErrors(hipblasSdgmm(handle, mode, m, n, x, lda, y, incx, z, ldc));
}

__global__ void elementWiseDivision(const float* x, const float* y, float* z,
                                    size_t size) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size) {
    z[idx] = x[idx] / y[idx];  // Perform element-wise division
  }
}

void CublasHandler::divide(const float* x, const float* y, float* z,
                           size_t size) {
  // Each thread processes one element; set the CUDA grid size
  int blockSize = 256;
  int gridSize = (size + blockSize - 1) / blockSize;

  // Launch the CUDA kernel to perform element-wise division
  elementWiseDivision<<<gridSize, blockSize>>>(x, y, z, size);

  // Check for CUDA errors
  checkCudaErrors(hipDeviceSynchronize());
}

// helper method for SubBackward
void CublasHandler::axpy(const float* x, float* y, float alpha, size_t size) {
    hipblasHandle_t handle = getInstance().getHandle();
    checkCublasErrors(hipblasSaxpy(handle, size, &alpha, x, 1, y, 1));
}



